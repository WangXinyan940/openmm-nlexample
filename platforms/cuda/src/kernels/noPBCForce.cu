#include "hip/hip_runtime.h"
extern "C" __global__ void calcTestForceNoPBC(
    mixed*              __restrict__     energyBuffer,
    real4*              __restrict__     posq,
    unsigned long long* __restrict__     forceBuffers,
    real*               __restrict__     params,
    int*                __restrict__     atomIndex,
    int*                __restrict__     pairidx0,
    int*                __restrict__     pairidx1,
    int                                  numParticles,
    int                                  paddedNumAtoms) {
    int totpair = numParticles * (numParticles - 1) / 2;
    for (int npair = blockIdx.x*blockDim.x+threadIdx.x; npair < totpair; npair += blockDim.x*gridDim.x) {
        int ii = pairidx0[npair];
        int jj = pairidx1[npair];
        real3 delta = make_real3(posq[jj].x-posq[ii].x,posq[jj].y-posq[ii].y,posq[jj].z-posq[ii].z);
        real R2 = delta.x * delta.x + delta.y * delta.y + delta.z * delta.z;
        real inverseR = RSQRT(R2);
        real p1p2 = params[atomIndex[ii]] * params[atomIndex[jj]];
        energyBuffer[npair] += p1p2 * inverseR * inverseR;
        real dEdRdR = - 2 * p1p2 * inverseR * inverseR * inverseR * inverseR;
        real3 force = dEdRdR * delta;
        atomicAdd(&forceBuffers[ii], static_cast<unsigned long long>((long long) (force.x*0x100000000)));
        atomicAdd(&forceBuffers[ii+paddedNumAtoms], static_cast<unsigned long long>((long long) (force.y*0x100000000)));
        atomicAdd(&forceBuffers[ii+2*paddedNumAtoms], static_cast<unsigned long long>((long long) (force.z*0x100000000)));
        atomicAdd(&forceBuffers[jj], static_cast<unsigned long long>((long long) (-force.x*0x100000000)));
        atomicAdd(&forceBuffers[jj+paddedNumAtoms], static_cast<unsigned long long>((long long) (-force.y*0x100000000)));
        atomicAdd(&forceBuffers[jj+2*paddedNumAtoms], static_cast<unsigned long long>((long long) (-force.z*0x100000000)));
    }
}

extern "C" __global__ void calcExcludeForceNoPBC(
    mixed*              __restrict__     energyBuffer,
    real4*              __restrict__     posq,
    unsigned long long* __restrict__     forceBuffers,
    real*               __restrict__     params,
    int*                __restrict__     atomIndex,
    int*                __restrict__     expairidx0,
    int*                __restrict__     expairidx1,
    int                                  totpair,
    int                                  numParticles,
    int                                  paddedNumAtoms) {
    for (int npair = blockIdx.x*blockDim.x+threadIdx.x; npair < totpair; npair += blockDim.x*gridDim.x) {
        int ii = expairidx0[npair];
        int jj = expairidx1[npair];
        real3 delta = make_real3(posq[jj].x-posq[ii].x,posq[jj].y-posq[ii].y,posq[jj].z-posq[ii].z);
        real R2 = delta.x * delta.x + delta.y * delta.y + delta.z * delta.z;
        real inverseR = RSQRT(R2);
        real p1p2 = params[atomIndex[ii]] * params[atomIndex[jj]];
        energyBuffer[npair] -= p1p2 * inverseR * inverseR;
        real dEdRdR = - 2 * p1p2 * inverseR * inverseR * inverseR * inverseR;
        real3 force = - dEdRdR * delta;
        atomicAdd(&forceBuffers[ii], static_cast<unsigned long long>((long long) (force.x*0x100000000)));
        atomicAdd(&forceBuffers[ii+paddedNumAtoms], static_cast<unsigned long long>((long long) (force.y*0x100000000)));
        atomicAdd(&forceBuffers[ii+2*paddedNumAtoms], static_cast<unsigned long long>((long long) (force.z*0x100000000)));
        atomicAdd(&forceBuffers[jj], static_cast<unsigned long long>((long long) (-force.x*0x100000000)));
        atomicAdd(&forceBuffers[jj+paddedNumAtoms], static_cast<unsigned long long>((long long) (-force.y*0x100000000)));
        atomicAdd(&forceBuffers[jj+2*paddedNumAtoms], static_cast<unsigned long long>((long long) (-force.z*0x100000000)));
    }
}